#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"
using namespace std;

#define BLK 16
#define STEP 4
#define VECTOR 16 

__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
#ifndef USE_CACHE
    const unsigned int bx = blockIdx.x;
    const unsigned int by = blockIdx.y; 
    const unsigned int tx = threadIdx.x; 
    const unsigned int ty = threadIdx.y;

    const unsigned int aBegin = N * (by * BLK); //A(0,by)
    const unsigned int aEnd = aBegin + N;
    const unsigned int aStep = BLK;             //offsetA
    
    const unsigned int bBegin = BLK * bx;       //B(bx,0)
    const unsigned int bStep = BLK * N;         //offsetB
    
    _DOUBLE_ cSub[BLK] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

    unsigned int a, b;

    for (a = aBegin,b = bBegin; a < aEnd - BLK; a += aStep,b += bStep) {
        __shared__ _DOUBLE_ As[BLK][BLK];
        __shared__ _DOUBLE_ Bs[BLK][BLK];
#pragma unroll
        for(int i = 0; i < BLK / STEP; i++) {
            As[ty + i * STEP][tx] = A[a + N * (ty + i * STEP) + tx];
            Bs[ty + i * STEP][tx] = B[b + N * (ty + i * STEP) + tx];
        }
        __syncthreads();
#pragma unroll
        for (int i = 0; i < BLK / STEP; i++) {
            for(int k = 0; k < BLK; k++) {
                cSub[i] += As[ty + i * STEP][k] * Bs[k][tx];
            }
        }

        __syncthreads();
    }

    // Maybe out of the matrix
    {
        __shared__ _DOUBLE_ As[BLK][BLK];
        __shared__ _DOUBLE_ Bs[BLK][BLK];
#pragma unroll
        for(int i = 0; i < BLK / STEP; i++) {
            if((ty + i * STEP + by * BLK < N) && ((a - aBegin) + tx < N)) As[ty + i * STEP][tx] = A[a + N * (ty + i * STEP) + tx];
            else As[ty + i * STEP][tx] = 0;
            if((ty + i * STEP + (a - aBegin) < N) && (BLK * bx + tx < N)) Bs[ty + i * STEP][tx] = B[b + N * (ty + i * STEP) + tx];
            else Bs[ty + i * STEP][tx] = 0;
        }
        __syncthreads();
#pragma unroll
        for (int i = 0; i < BLK / STEP; i++) {
            for(int k = 0; k < BLK; k++) {
                cSub[i] += As[ty + i * STEP][k] * Bs[k][tx];
            }
        }

        __syncthreads();
    }

    for(int i = 0; i < BLK / STEP; i++) {
        if(by * BLK + ty  + i * STEP < N && bx * BLK + tx < N) {
            int cIndex = (by * BLK + ty + i * STEP) * N + (bx * BLK + tx);
            C[cIndex] = cSub[i];
        }
    }
#else
    const int I =  blockIdx.x * blockDim.x + threadIdx.x;
    const int J =  (blockIdx.y * blockDim.y + threadIdx.y) * VECTOR;
    _DOUBLE_ _c[VECTOR] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    if(I < N) {
        if(N - J >= VECTOR) {
#pragma unroll
            for (int k = 0; k < N; k++) {
                _DOUBLE_ a = A[I * N + k];
                for(int i = 0; i < VECTOR; i++) {
                    _DOUBLE_ b = B[k * N + J + i];
                    _c[i] += a * b;
                }
            }
#pragma unroll
            for(int i = 0; i < VECTOR; i++) {
                C[I * N + J + i] = _c[i];
            }
        }
        else {
            int upBound =  N - J;
#pragma unroll
            for (int k = 0; k < N; k++) {
                _DOUBLE_ a = A[I * N + k];
                for(int i = 0; i < upBound; i++) {
                    _DOUBLE_ b = B[k * N + J + i];
                    _c[i] += a * b;
                }
            }
#pragma unroll
            for(int i = 0; i < upBound; i++) {
                C[I * N + J + i] = _c[i];
            }
        }
    }
#endif
}
