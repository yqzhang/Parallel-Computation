
#include <hip/hip_runtime.h>
#define BLK 16
#define STEP 4
#define VECTOR 16

void setGrid(int n, dim3 &blockDim, dim3 &gridDim)
{
#ifndef USE_CACHE
   // set your block dimensions and grid dimensions here
   blockDim.x = BLK;
   blockDim.y = STEP;
   gridDim.x = (n + BLK - 1) / BLK;
   gridDim.y = (n + BLK - 1) / BLK;
#else
   blockDim.x = VECTOR * 2;
   blockDim.y = 2;
   gridDim.x = (n + 2 * VECTOR - 1) / (2 * VECTOR);
   gridDim.y = (n + 2 * VECTOR - 1) / (2 * VECTOR);
#endif
}

